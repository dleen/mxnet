#include "hip/hip_runtime.h"
#include "./hadamard_sparse_op.h"
#include "./elemwise_binary_op.h"
#include "./elemwise_binary_broadcast_op.h"
#include <mshadow/tensor.h>

#define WARPS_PER_BLOCK 1
#define THREADS_PER_BLOCK 256
#define ELEMENTARY_LOG2SIZE 11

#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)


namespace mshadow {
namespace cuda {


template <typename DType>
__global__ void hadamard_sparse_forward_kernel(const int nthreads, DType *out, DType *indices, DType *value, DType *key, int in_dim, int out_dim) {


   const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= nthreads){
         return;
     }

    int k = out_dim;
    int col = index;
    int nnz = in_dim;
    DType *pValues = value;
    DType *pKeys = key;
    DType *pIndices = indices;

    for (int j = nnz; j; j--) {

            int ind = (int) *(pIndices+col);
            int row = (int) *pKeys;
            int keyvalue = (int) *(pKeys+1);
            DType *pRes = out;
            pRes += row*k+col;
            *pRes += ((__popcll(ind & keyvalue) & 1) * -2 + 1) * (*pValues);

            pKeys+=2;
            pValues++;

    }
}


template <typename DType>
inline void hadamardTransformGSparse(Tensor<gpu, 2, DType> &out, Tensor<gpu, 1, DType> &value, Tensor<gpu, 2, DType> &key, Tensor<gpu, 1, DType> &indices) {

    int in_dim = (unsigned int) key.shape_[0];
    int n_samples = (unsigned int) out.shape_[0];
    int out_dim = (unsigned int) indices.shape_[1];
    out = 0;
    DType *out_p = out.dptr_;
    DType *value_p = value.dptr_;
    DType *key_p = key.dptr_;

    DType *indices_p = indices.dptr_;
    int processing_batch_size = 2<<12;
    LOG(INFO)<<processing_batch_size;
    int upper_bound = in_dim/processing_batch_size;
    if (in_dim%processing_batch_size == 0){
      upper_bound = upper_bound-1;
    }
    upper_bound = upper_bound>0? upper_bound:0;

    int bstart = 0;
    for ( int i = 0; i <= upper_bound; i++ ){
        int batchlen = min(processing_batch_size, in_dim - bstart );
        int threads_per_block = min(THREADS_PER_BLOCK, batchlen);
        int nblocks = (batchlen + threads_per_block - 1) / threads_per_block ;

        hadamard_sparse_forward_kernel<DType><<<nblocks, threads_per_block>>>(out_dim, out_p, indices_p, value_p+bstart, key_p+bstart*2, batchlen, out_dim);
        bstart = (i+1)*batchlen;


    }
}

}
}


namespace mxnet {
namespace op {


template<typename xpu>
void hadamardTransformGeneralSparse(const nnvm::NodeAttrs& attrs,
                       const OpContext& ctx,
                       const std::vector<TBlob>& inputs,
                       const std::vector<OpReqType>& req,
                       const std::vector<TBlob>& outputs) {
    using namespace mshadow;
    using namespace mshadow::expr;

    CHECK_EQ(inputs.size(), 3);
    CHECK_EQ(outputs.size(), 1);
    Stream<xpu> *s = ctx.get_stream<xpu>();

    MSHADOW_TYPE_SWITCH(outputs[0].type_flag_, DType, {

            Tensor<xpu, 2, DType> out = outputs[0].FlatTo2D<xpu, DType>(s);
            Tensor<xpu, 2, DType> key = inputs[0].FlatTo2D<xpu, DType>(s);
            Tensor<xpu, 1, DType> value = inputs[1].FlatTo1D<xpu, DType>(s);
            Tensor<xpu, 1, DType> indices = inputs[2].FlatTo1D<xpu, DType>(s);

            mshadow::cuda::hadamardTransformGSparse<DType>(out, value, key,  indices);


    });
}


NNVM_REGISTER_OP(sparse_inplace)
.set_attr<FCompute>("FCompute<gpu>", hadamardTransformGeneralSparse<gpu>);

NNVM_REGISTER_OP(_backward_sparse_inplace)
.set_attr<FCompute>("FCompute<gpu>", hadamardTransformGeneralSparse<gpu>);

}
}